#include "hip/hip_runtime.h"



//Include
//-----------------------------------------------
#include "rbRollCUDAFunctionality.h"




//Kernels
//-----------------------------------------------
namespace kernels{

	__global__ void modifyArray(float* d_arrayToModify, float* d_resultArrayToModify)
	{
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		d_resultArrayToModify[index] = d_arrayToModify[index] * 2.0;
	};


	__global__ void vectorMatrixMultiplication(float* vertexPositionXArray, float* vertexPositionYArray, float* vertexPositionZArray, float* rotationMatrixArray, float* resultVertexPositionXArray, float* resultVertexPositionYArray, float* resultVertexPositionZArray, int numberOperations)
	{

		int index = blockIdx.x * blockDim.x + threadIdx.x;
		
		if(index <= numberOperations)
		{
			resultVertexPositionXArray[index] = (vertexPositionXArray[index] * rotationMatrixArray[0]) + (vertexPositionYArray[index] * rotationMatrixArray[1]) + (vertexPositionZArray[index] * rotationMatrixArray[2]);
			resultVertexPositionYArray[index] = (vertexPositionXArray[index] * rotationMatrixArray[4]) + (vertexPositionYArray[index] * rotationMatrixArray[5]) + (vertexPositionZArray[index] * rotationMatrixArray[6]);
			resultVertexPositionZArray[index] = (vertexPositionXArray[index] * rotationMatrixArray[8]) + (vertexPositionYArray[index] * rotationMatrixArray[9]) + (vertexPositionZArray[index] * rotationMatrixArray[10]);
		}
	};
	
	
	__global__ void addTwoIntegers(int* a, int* b, int* result)
	{
		*result = *a+*b;
	};

}



//Definitions
//-----------------------------------------------
void cuda::executeModifyArray(float* arrayToModify, int arrayLength)
{

	//byteSize
	int byteSize = arrayLength * sizeof(float);

	//d_arrayToModify, d_resultArrayToModify
	float* d_arrayToModify, *d_resultArrayToModify;

	//allocate mem on GPU
	hipMalloc((void**)&d_arrayToModify, byteSize);
	hipMalloc((void**)&d_resultArrayToModify, byteSize);

	//copy host bytes to gpu
	hipMemcpy(d_arrayToModify, arrayToModify, byteSize, hipMemcpyHostToDevice);

	//launch kernel
	int blockDim = arrayLength;
	int numberOfBlocks = 1;
	kernels::modifyArray<<<numberOfBlocks, blockDim>>>(d_arrayToModify, d_resultArrayToModify);

	//copy results from device to host
	hipMemcpy(arrayToModify, d_resultArrayToModify, byteSize, hipMemcpyDeviceToHost);

	//free pointer
	hipFree(d_arrayToModify); hipFree(d_resultArrayToModify);

}

void cuda::executeVectorMatrixMultiplication(float* vertexPositionXArray, float* vertexPositionYArray, float* vertexPositionZArray, float* rotationMatrixArray, int vertexPositionArrayLength)
{

	
	//byteSize
	int byteSizeVertexPositionArray = vertexPositionArrayLength * sizeof(float);
	int byteSizeRotationMatrixArray = 16 * sizeof(float);

	//device pointers for vertexPositionArrays
	float* d_vertexPositionXArray, *d_vertexPositionYArray, *d_vertexPositionZArray;
	//device pointers for result arrays
	float* d_resultVertexPositionXArray, *d_resultVertexPositionYArray, *d_resultVertexPositionZArray;
	//device pointer rotationMatrixArray
	float* d_rotationMatrixArray;

	//allocate memory on device
	hipMalloc((void**)&d_vertexPositionXArray, byteSizeVertexPositionArray);
	hipMalloc((void**)&d_vertexPositionYArray, byteSizeVertexPositionArray);
	hipMalloc((void**)&d_vertexPositionZArray, byteSizeVertexPositionArray);

	hipMalloc((void**)&d_resultVertexPositionXArray, byteSizeVertexPositionArray);
	hipMalloc((void**)&d_resultVertexPositionYArray, byteSizeVertexPositionArray);
	hipMalloc((void**)&d_resultVertexPositionZArray, byteSizeVertexPositionArray);
	
	hipMalloc((void**)&d_rotationMatrixArray, byteSizeRotationMatrixArray);

	//copy values from host to device
	hipMemcpy(d_vertexPositionXArray, vertexPositionXArray, byteSizeVertexPositionArray, hipMemcpyHostToDevice);
	hipMemcpy(d_vertexPositionYArray, vertexPositionYArray, byteSizeVertexPositionArray, hipMemcpyHostToDevice);
	hipMemcpy(d_vertexPositionZArray, vertexPositionZArray, byteSizeVertexPositionArray, hipMemcpyHostToDevice);

	hipMemcpy(d_rotationMatrixArray, rotationMatrixArray, byteSizeRotationMatrixArray, hipMemcpyHostToDevice);

	//execute kernel
	int blockSize = 512;
	int numberOfBlocks =  (vertexPositionArrayLength / blockSize) + 1;

	kernels::vectorMatrixMultiplication<<<numberOfBlocks, blockSize>>>(d_vertexPositionXArray, d_vertexPositionYArray, d_vertexPositionZArray, d_rotationMatrixArray, d_resultVertexPositionXArray, d_resultVertexPositionYArray, d_resultVertexPositionZArray, vertexPositionArrayLength);

	//Copy values back to host
	hipMemcpy(vertexPositionXArray, d_resultVertexPositionXArray, byteSizeVertexPositionArray, hipMemcpyDeviceToHost);
	hipMemcpy(vertexPositionYArray, d_resultVertexPositionYArray, byteSizeVertexPositionArray, hipMemcpyDeviceToHost);
	hipMemcpy(vertexPositionZArray, d_resultVertexPositionZArray, byteSizeVertexPositionArray, hipMemcpyDeviceToHost);

	//free device pointer
	hipFree(d_vertexPositionXArray); hipFree(d_vertexPositionYArray); hipFree(d_vertexPositionZArray);
	hipFree(d_resultVertexPositionXArray);hipFree(d_resultVertexPositionYArray);hipFree(d_resultVertexPositionZArray);
	hipFree(d_rotationMatrixArray);
	
}


int cuda::executeAddTwoIntegers(int paramA, int paramB)
{
	
	//byteSize
	int byteSize = sizeof(int);

	//initialze integers
	int a = paramA, b = paramB, result;

	//device pointers for integers
	int* d_a, *d_b, *d_result;
	
	//allocate memory on the device
	hipMalloc((void**)&d_a, byteSize);
	hipMalloc((void**)&d_b, byteSize);
	hipMalloc((void**)&d_result, byteSize);

	//copy values from host to device
	hipMemcpy(d_a, &a, byteSize, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, byteSize, hipMemcpyHostToDevice);
	
	//execute kernel
	kernels::addTwoIntegers<<<1,1>>>(d_a, d_b, d_result);

	//copy results back from device to host
	hipMemcpy(&result, d_result, byteSize, hipMemcpyDeviceToHost);

	//CleanUp dynamically allocated memory
	hipFree(d_a);hipFree(d_b);hipFree(d_result);

	//printf("Result: %d\n" , result);
	//printf("Cuda Error Msg: %s\n", hipGetErrorString(hipGetLastError()));
	
	return result;
}


int cuda::executeDeviceQuery(void)
{
	//deviceCount
	int deviceCount = 0;
	hipError_t cudaErrorId;

	cudaErrorId = hipGetDeviceCount(&deviceCount);
	//printf("DeviceCount: %d\n", deviceCount);
	//printf("Cuda ErrorId: %d\nErrorMsg: %s\n", int(cudaErrorId), hipGetErrorString(cudaErrorId));

	return deviceCount;
}